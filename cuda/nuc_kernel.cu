#include "hip/hip_runtime.h"
#include "nuc_kernel.h"

#define THREADS 512

__global__ void nuc(int *out, float *gain, float *offset, int size) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i<size) out[i] =(int)gain[i]*(out[i] - offset[i]);
}


void nucCaller(int *out, float *gain, float *offset, int n){
    // copy offset to device
    float *d_offset = NULL;
    hipMalloc((void **)&d_offset, n*n*sizeof(float));
    hipMemcpy(d_offset, offset, n*n*sizeof(float), hipMemcpyHostToDevice);
    // copy gain to device
    float *d_gain = NULL;
    hipMalloc((void **)&d_gain, n*n*sizeof(float));
    hipMemcpy(d_gain, gain, n*n*sizeof(float), hipMemcpyHostToDevice);

    int *d_out = NULL;
    hipMalloc((void **)&d_out, n*n*sizeof(int));
    hipMemcpy(d_out, out, n*n*sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = THREADS;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;

    nuc<<<blocksPerGrid,threadsPerBlock>>>(d_out, d_gain, d_offset,n);
    hipMemcpy(out, d_out, n*n*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_out);
    hipFree(d_offset);
    hipFree(d_gain);

    return;
};
