#include "hip/hip_runtime.h"
#include "nuc_kernel.h"
#include <stdio.h>

#define THREADS 256

__global__ void nuc(uint16_t *out,const float *gain,const float *offset, int size) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i<size)	out[i] = (uint16_t)(gain[i]*(out[i] - offset[i]));
}


void nucCaller(uint16_t *out, float *gain, float *offset, int n){
    // copy offset to device
    float *d_offset = NULL;
    hipMalloc((void **)&d_offset, n*sizeof(float));
    hipMemcpy(d_offset, offset, n*sizeof(float), hipMemcpyHostToDevice);
    // copy gain to device
    float *d_gain = NULL;
    hipMalloc((void **)&d_gain, n*sizeof(float));
    hipMemcpy(d_gain, gain, n*sizeof(float), hipMemcpyHostToDevice);
    uint16_t *d_out = NULL;
    hipMalloc((void **)&d_out, n*sizeof(uint16_t));
    hipMemcpy(d_out, out, n*sizeof(uint16_t), hipMemcpyHostToDevice);
    int threadsPerBlock = THREADS;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;

    nuc<<<blocksPerGrid,threadsPerBlock>>>(d_out, d_gain, d_offset,n);
    hipMemcpy(out, d_out, n*sizeof(uint16_t), hipMemcpyDeviceToHost);

    hipFree(d_out);
    hipFree(d_offset);
    hipFree(d_gain);

    return;
};
