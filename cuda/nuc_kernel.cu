#include "hip/hip_runtime.h"
#include "nuc_kernel.h"
#include <stdio.h>

#define THREADS 256

__global__ void nuc(int *out,const float *gain,const float *offset, int size) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i<size)	out[i] = (int)(gain[i]*(out[i] - offset[i]));
}


void nucCaller(int *out, float *gain, float *offset, int n){
    // copy offset to device
    float *d_offset = NULL;
    hipMalloc((void **)&d_offset, n*sizeof(float));
    hipMemcpy(d_offset, offset, n*sizeof(float), hipMemcpyHostToDevice);
    // copy gain to device
    float *d_gain = NULL;
    hipMalloc((void **)&d_gain, n*sizeof(float));
    hipMemcpy(d_gain, gain, n*sizeof(float), hipMemcpyHostToDevice);
    int *d_out = NULL;
    hipMalloc((void **)&d_out, n*sizeof(int));
    hipMemcpy(d_out, out, n*sizeof(int), hipMemcpyHostToDevice);
    int threadsPerBlock = THREADS;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;

    printf("%d, %d, %d, %d, %f, %f\n",
	   out[0], out[1], out[2], out[3], offset[4], gain[5]);

    nuc<<<blocksPerGrid,threadsPerBlock>>>(d_out, d_gain, d_offset,n);
    hipMemcpy(out, d_out, n*sizeof(int), hipMemcpyDeviceToHost);

    printf("%d, %d, %d, %d, %f, %f\n",
	   out[0], out[1], out[2], out[3], offset[4], gain[5]);
    hipFree(d_out);
    hipFree(d_offset);
    hipFree(d_gain);

    return;
};
